#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <algorithm>
#include <hipcub/hipcub.hpp>

void checkCuda(hipError_t result, const char *func){
    if (result != hipSuccess){
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        exit(-1);
    }
}

#define CUDA_CHECK(ans) { checkCuda((ans), #ans); }

__global__ void checkIfArrayIsSafe(int** arrays, int* is_safe_array, int n, int* len_of_arrays){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    bool is_increasing = false;

    if (idx >= n){
        return;
    }
    if (arrays[idx][0] < arrays[idx][1]){
        is_increasing = true;
    }
    
    for(int i = 1; i < len_of_arrays[idx]; i++){
        if(arrays[idx][i-1] == arrays[idx][i]){
            is_safe_array[idx] = 0;
            return;
        }
        else if(is_increasing && (arrays[idx][i-1] > arrays[idx][i])){
            is_safe_array[idx] = 0;
            return;
        }
        else if (!is_increasing && (arrays[idx][i-1] < arrays[idx][i])){
            is_safe_array[idx] = 0;
            return;
        }
        else if((std::abs(arrays[idx][i-1] - arrays[idx][i]) > 3) || 
                (std::abs(arrays[idx][i-1] - arrays[idx][i]) < 1)){
            is_safe_array[idx] = 0;
            return;
        }
    }
    is_safe_array[idx] = 1;
}

// move get safe array code to a function
int* get_is_safe_array(int** arrays, int* len_of_arrays, int n){
    // move array to device
    int** d_arrays;
    int* d_len_of_arrays;

    // allocate memory for the array of pointers, which have dynamically allocated memory
    CUDA_CHECK(hipMalloc(&d_arrays, n * sizeof(int*)));
    // CUDA_CHECK(hipMemcpy(d_arrays, arrays, n * sizeof(int*), hipMemcpyHostToDevice));
    for (int i = 0; i < n; i++){
        int* d_array;
        CUDA_CHECK(hipMalloc(&d_array, len_of_arrays[i] * sizeof(int)));
        CUDA_CHECK(hipMemcpy(d_array, arrays[i], len_of_arrays[i] * sizeof(int), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(&d_arrays[i], &d_array, sizeof(int*), hipMemcpyHostToDevice));
    }

    CUDA_CHECK(hipMalloc(&d_len_of_arrays, n * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_len_of_arrays, len_of_arrays, n * sizeof(int), hipMemcpyHostToDevice));

    // allocate memory for the result
    int* is_safe_array = new int[n];
    int* d_is_safe_array;
    CUDA_CHECK(hipMalloc(&d_is_safe_array, n * sizeof(int)));
    CUDA_CHECK(hipMemset(d_is_safe_array, 0, n * sizeof(int)));

    // calculate the number of blocks and threads
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // run the kernel
    checkIfArrayIsSafe<<<numBlocks, blockSize>>>(d_arrays, d_is_safe_array, n, d_len_of_arrays);

    // run debug
    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // copy the result back
    CUDA_CHECK(hipMemcpy(is_safe_array, d_is_safe_array, n * sizeof(int), hipMemcpyDeviceToHost));

    return is_safe_array;
}


__global__ void sumArray(int* d_array, int n, int *result){
    extern __shared__ int shared_data[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tix = threadIdx.x;

    // if (idx < n) {
    //     printf("1st try: Block %d, Thread %d: d_array[%d] = %d\n", blockIdx.x, tix, idx, d_array[idx]);
    // }
    if(tix < blockDim.x && idx >= n){
        return;
    }
    // Load elements into shared memory
    if (idx < n){
        shared_data[tix] = d_array[idx];
    }
    else{
        shared_data[tix] = 0;
    }

    // Synchronize threads to make sure all elements are loaded
    __syncthreads(); 

    // if (idx < n) {
    //     printf("2nd try: Block %d, Thread %d: d_array[%d] = %d\n", blockIdx.x, tix, idx, d_array[idx]);
    // }
    // Perform the reduction
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tix < stride) {
            shared_data[tix] += shared_data[tix + stride];
        }
        __syncthreads();
    }

    if(tix == 0){
        atomicAdd(result, shared_data[0]);
    }
}

int main(int argc, char** argv){
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    std::cout << "Number of CUDA devices: " << deviceCount << std::endl;

    // open the file
    std::cout << "Reading file: " << argv[1] << std::endl;
    if (argc < 2) {
        std::cerr << "Usage: " << argv[1] << " <input_file_path>" << std::endl;
        return 1;
    }
    std::cout << "Reading file: " << argv[1] << std::endl;
    std::string filePath = argv[1];
    std::ifstream file(filePath);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open file: " << filePath << std::endl;
        return 1;
    }
    
    // read the number of elements

    // Use istreambuf_iterator to count newlines
    int n = std::count(
        std::istreambuf_iterator<char>(file),
        std::istreambuf_iterator<char>(),
        '\n'
    );

    std::cout << "Number of lines: " << n << std::endl;

    // return file pointer to the beginning
    file.close();
    file.open(filePath);

    // make an array of arrays
    int** arrays = new int*[n];
    int* len_of_arrays = new int[n];

    for (int i = 0; i < n; i++){
        std::string read_line;
        std::getline(file, read_line);
        std::string delimiter = " ";

        // calculate number of elements in the line
        int num_elements = std::count(
            read_line.begin(),
            read_line.end(),
            ' '
        ) + 1;

        // allocate memory for the array
        arrays[i] = new int[num_elements];

        // read the elements of the array
        int pos;
        for (int j = 0; j < num_elements; j++){
            pos = read_line.find(delimiter);
            arrays[i][j] = std::stoi(read_line.substr(0, pos));
            read_line.erase(0, pos + delimiter.length());
        }

        len_of_arrays[i] = num_elements;
    }

    // close the file
    file.close();

    int* is_safe_array = get_is_safe_array(arrays, len_of_arrays, n);



    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    int how_many_safe = 0;
    int sharedMemSize = blockSize * sizeof(int);
    int* d_how_many_safe;
    int* d_is_safe_array;

    CUDA_CHECK(hipMalloc(&d_is_safe_array, n * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_is_safe_array, is_safe_array, n * sizeof(int), hipMemcpyHostToDevice));


    CUDA_CHECK(hipMalloc(&d_how_many_safe, sizeof(int)));
    CUDA_CHECK(hipMemset(d_how_many_safe, 0, sizeof(int)));


    sumArray<<<numBlocks, blockSize, sharedMemSize>>>(d_is_safe_array, n, d_how_many_safe);

    // Move back to device
    CUDA_CHECK(hipMemcpy(&how_many_safe, d_how_many_safe, sizeof(int), hipMemcpyDeviceToHost));

    std::cout << "Number of safe arrays: " << how_many_safe << std::endl;


}